#include "hip/hip_runtime.h"
#include "nbody.h"

#include <math.h>

#include <iostream>

using namespace std;

/**
 * Update the acceleration array and reset collided boolean for the
 * CUDA implementation.
 * 
 * @param acceleration array of double of the bodies' 3D acceleration
 * @param positions array of double of the bodies' 3D positions
 * @param masses array of double of bodies' masses
 * @param collided array of boolean of whether a body collided this second
 * @param num_bodies the number of bodies in simulation
 */
__global__ void update_acceleration_and_reset_collided_cuda(double* const acceleration, 
    double* const positions, double* const masses, bool* const collided, 
    const int num_bodies) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < num_bodies) {
        acceleration[index * 3] = 0;
        acceleration[index * 3 + 1] = 0;
        acceleration[index * 3 + 2] = 0;
        collided[index] = false;

        for (int i = 0; i < num_bodies; i++) {
            if (i != index) {
                const double d = sqrt(pow(positions[index * 3] - positions[i * 3], 2) 
                    + pow(positions[index * 3 + 1] - positions[i * 3 + 1], 2)
                    + pow(positions[index * 3 + 2] - positions[i * 3 + 2], 2));
                const double g = (masses[index] * G) / pow(d, 3);

                acceleration[index * 3] += g * (positions[i * 3] - positions[index * 3]);
                acceleration[index * 3 + 1] += g * (positions[i * 3 + 1] - 
                    positions[index * 3 + 1]);
                acceleration[index * 3 + 2] += g * (positions[i * 3 + 2] - 
                    positions[index * 3 + 2]);
            }
        }
    }
}

/**
 * Calculates the new velocity and location of the bodies for the CUDA implementation.
 * 
 * @param positions array of double of the bodies' 3D positions
 * @param velocities array of double of the bodies' 3D velocities
 * @param acceleration array of double of the bodies' 3D acceleration
 * @param collided array of boolean of whether a body collided this second
 * @param num_bodies the number of bodies in simulation
 */
__global__ void update_velocity_and_location_cuda(double* const positions, double* const velocities, 
    double* const acceleration, bool* const collided, const int num_bodies) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < num_bodies) {
        if (!collided[index]) {
            velocities[index * 3] += acceleration[index * 3];
            velocities[index * 3 + 1] += acceleration[index * 3 + 1];
            velocities[index * 3 + 2] += acceleration[index * 3 + 2];
        }
        positions[index * 3] += velocities[index * 3];
        positions[index * 3 + 1] += velocities[index * 3 + 1];
        positions[index * 3 + 2] += velocities[index * 3 + 2];
    }
}


/**
 * Handles elastic collisions for the CUDA implementation.
 * 
 * @param velocities array of double of the bodies' 3D velocities
 * @param positions array of double of the bodies' 3D positions
 * @param masses array of double of bodies' masses
 * @param radii array of double of bodies' radii
 * @param collided array of boolean of whether a body collided this second
 * @param num_bodies the number of bodies in simulation
 * 
 */
void handle_collisions_cuda(double* const velocities, double* const positions, 
    double* const masses, double* const radii, bool* const collided, const int num_bodies) {
    for (int i = 0; i < num_bodies; i++) {
        for (int j = 0; j < i; j++) {
            if (sqrt(pow(positions[i * 3] - positions[j * 3], 2) + pow(positions[i * 3 + 1] - 
                positions[j * 3 + 1], 2) + pow(positions[i * 3 + 2] - positions[j * 3 + 2], 2)) 
                < (radii[i] + radii[j])) {
                collided[i] = true;
                collided[j] = true;

                const double k1 = (2 * masses[i]) / (masses[i] + masses[j]);
                const double k2 = (masses[i] -  masses[j]) / (masses[i] + masses[j]);
                const double k3 = (2 * masses[j]) / (masses[i] + masses[j]);

                double tmpvel_ = velocities[i * 3];
                velocities[i * 3] = (k2 * tmpvel_) + (k3 * velocities[j * 3]);
                velocities[j * 3] = (k1 * tmpvel_) - (k2 * velocities[j * 3]);

                tmpvel_ = velocities[i * 3 + 1];
                velocities[i * 3 + 1] = (k2 * tmpvel_) + (k3 * velocities[j * 3 + 1]);
                velocities[j * 3 + 1] = (k1 * tmpvel_) - (k2 * velocities[j * 3 + 1]);

                tmpvel_ = velocities[i * 3 + 2];
                velocities[i * 3 + 2] = (k2 * tmpvel_) + (k3 * velocities[j * 3 + 2]);
                velocities[j * 3 + 2] = (k1 * tmpvel_) - (k2 * velocities[j * 3 + 2]);
            }
        }
    }
}

/**
 * Print the current result of the simulation for the CUDA implementation.
 * 
 * @param second current second of simulation
 * @param num_bodies the number of bodies in simulation
 * @param positions array of double of the bodies' 3D positions
 * @param velocities array of double of the bodies' 3D velocities
 * @param acceleration array of double of the bodies' 3D acceleration
 */
void output_result_cuda(const int& second, const int num_bodies, 
    const double* const positions, const double* const velocities, 
    const double* const acceleration) {
    cout << "Second: " << second << "\n";
    for (int i = 0; i < num_bodies; i++) {
        cout << "Body: " << i << "\n";
        cout << "Position: (" << positions[i * 3] << ", " << positions[i * 3 + 1] 
            << ", " << positions[i * 3 + 2] << ")\n";
        cout << "Velocity: (" << velocities[i * 3] << ", " << velocities[i * 3 + 1] 
            << ", " << velocities[i * 3 + 2] << ")\n";
        cout << "Acceleration: (" << acceleration[i * 3] << ", " << 
            acceleration[i * 3 + 1] << ", " << acceleration[i * 3 + 2] << ")\n";
    }
    cout << "\n";
}

void nbody_cuda(const vector<Body>& bodies, const int& seconds, const bool& output) {
    // Prepare data for GPU
    double *positions, *velocities, *acceleration, *masses, *radii;
    bool *collided;
    hipMallocManaged(&positions, bodies.size() * sizeof(struct Vector3D));
    hipMallocManaged(&velocities, bodies.size() * sizeof(struct Vector3D));
    hipMallocManaged(&acceleration, bodies.size() * sizeof(struct Vector3D));
    hipMallocManaged(&masses, bodies.size() * sizeof(double));
    hipMallocManaged(&radii, bodies.size() * sizeof(double));
    hipMallocManaged(&collided, bodies.size() * sizeof(bool));

    for (int i = 0; i < bodies.size(); i++) {
        positions[i * 3] = bodies[i].pos_.x_;
        positions[i * 3 + 1] = bodies[i].pos_.y_;
        positions[i * 3 + 2] = bodies[i].pos_.z_;
        velocities[i * 3] = bodies[i].vel_.x_;
        velocities[i * 3 + 1] = bodies[i].vel_.y_;
        velocities[i * 3 + 2] = bodies[i].vel_.z_;
        acceleration[i * 3] = bodies[i].acc_.x_;
        acceleration[i * 3 + 1] = bodies[i].acc_.y_;
        acceleration[i * 3 + 2] = bodies[i].acc_.z_;

        masses[i] = bodies[i].mass_;
        radii[i] = bodies[i].radius_;
        collided[i] = bodies[i].collided_;
    }
    
    for (int s = 0; s < seconds; s++) {
        if (output) {
            output_result_cuda(s, bodies.size(), positions, velocities, acceleration);
        }

        update_acceleration_and_reset_collided_cuda<<<(bodies.size() + THREADS_PER_BLOCK - 1) 
            / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(acceleration, positions, masses, collided, 
            bodies.size());
        hipDeviceSynchronize();

        handle_collisions_cuda(velocities, positions, masses, radii, collided, bodies.size());

        update_velocity_and_location_cuda<<<(bodies.size() + THREADS_PER_BLOCK - 1) 
            / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(positions, velocities, acceleration, collided, 
            bodies.size());
        hipDeviceSynchronize();
    }

    if (output) {
        output_result_cuda(seconds, bodies.size(), positions, velocities, acceleration);
    }

    hipFree(positions);
    hipFree(velocities);
    hipFree(acceleration);
    hipFree(masses);
    hipFree(radii);
    hipFree(collided);
}
